#include "hip/hip_runtime.h"
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
#include <stdio.h>
#include <sys/time.h>
#include <string>
#include "arrayMultCUDA.cuh"

void check_result(hipError_t code, const char* message)
{
  if (code != hipSuccess)
    {
      fprintf(stderr, "%s (%d): %s\n", message, (int) code, 
	      hipGetErrorString(code));
      exit(-1);
    }
}

// Forward declaration of the matrix addition kernel
__global__ void MatAddKernel(const MatrixForCUDA, const MatrixForCUDA, MatrixForCUDA);

// MatrixForCUDA add - Host code
// MatrixForCUDA dimensions are assumed to be multiples of BLOCK_SIZE
void MatAdd(const MatrixForCUDA A, const MatrixForCUDA B, MatrixForCUDA C, int block_size, 
	    dim3 dimGrid, dim3 dimBlock, std::shared_ptr<LPISupport::SampleCollection> pSamples, unsigned int numberOfIterations)
{
    // Load A and B to device memory
    MatrixForCUDA d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    check_result(hipMalloc(&d_A.elements, size),
		 "Unable to allocate device memory");
    check_result(hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice),
		 "Unable to copy variable to device");
    MatrixForCUDA d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    check_result(hipMalloc(&d_B.elements, size),
		 "Unable to allocate device memory");
    check_result(hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice),
		 "Unable to copy variable to device");

    // Allocate C in device memory
    MatrixForCUDA d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    check_result(hipMalloc(&d_C.elements, size),
		 "Unable to allocate device memory");

    // Invoke kernel: Mal dimGrid si B.width y A.height no son múltiplos de 
    // block_size
    // dim3 dimBlock(block_size, block_size);
    // dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (unsigned int iteration = 0; iteration < numberOfIterations; iteration++) {
        cout << "Iteration #" << iteration << std::endl;
        //gettimeofday(&t1, NULL);
        hipEventRecord(start, 0);
        MatAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float gpuElapsedTime;
        hipEventElapsedTime(&gpuElapsedTime, start, stop); // in ms
        /*  ((t2.tv_sec - t1.tv_sec) * 1000.0) +
        ((t2.tv_usec - t1.tv_usec) / 1000.0); */
        pSamples->appendSample(gpuElapsedTime / 1000);
        //gettimeofday(&t2, NULL);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);  

    // Read C from device memory
    check_result(hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost),
		 "Unable to copy output variable from device");

    // Free device memory
    check_result(hipFree(d_A.elements), "Error freeing device memory");
    check_result(hipFree(d_B.elements), "Error freeing device memory");
    check_result(hipFree(d_C.elements), "Error freeing device memory");
}

// MatrixForCUDA add kernel called by MatMul()
__global__ void MatAddKernel(MatrixForCUDA A, MatrixForCUDA B, MatrixForCUDA C)
{
    // Each thread computes one element of C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row > A.height-1) /* Nos salimos del rango de índices válidos */
      return; 
    if(A.height <= 32) // Son demasiados índices a mostrar con N grande
      printf("threadIdx.y: %d, blockIdx.y: %d, row: %d\n", 
	     threadIdx.y, blockIdx.y, row); 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col > B.width-1) /* Nos salimos del rango de índices válidos */
      return; 
    if(B.width <= 32) // Son demasiados índices a mostrar con N grande
      printf("threadIdx.x: %d, blockIdx.x: %d, col: %d\n", 
	     threadIdx.x, blockIdx.x, col); 

        C.elements[row * C.width + col] = A.elements[row * A.width + col] + B.elements[row * B.width + col];
}

